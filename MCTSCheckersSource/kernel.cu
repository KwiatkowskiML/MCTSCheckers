﻿#include <stdio.h>
#include "Board.h"
#include "MoveGenerator.h"
#include "CheckersTestSuite.h"
#include "PlayerCPU.h"
#include "PlayerGPU.cuh"
#include "Game.h"
#include <random>

void CompareSimulations(Board board, int times, PieceColor playerColor, PieceColor playerToMove)
{
	int resultcpu = 0;
	int resultgpu = 0;

    for (int i = 0; i < times; i++)
    {
		int winner = board.simulateGame(playerToMove);
        int result = 0;

        if (winner == BLACK_WIN && playerColor == PieceColor::Black)
            result = WIN;
        else if (winner == WHITE_WIN && playerColor == PieceColor::White)
            result = WIN;
        else if (winner == DRAW)
            result = DRAW;
        else
            result = LOOSE;

		resultcpu += result;
    }

    for (int i = 0; i < times; i++)
    {
        std::random_device rd; // Seed
        std::mt19937 gen(rd()); // Mersenne Twister engine
        std::uniform_int_distribution<> dist(-999999, 999999);

		int z = dist(gen);
		int w = dist(gen);
		int jsr = dist(gen);
		int jcong = dist(gen);

		int winner = simulateGameGpu(board.getWhitePawns(), board.getBlackPawns(), board.getKings(), playerToMove, z, w, jsr, jcong);

		int result = 0;
		if (winner == BLACK_WIN && playerColor == PieceColor::Black)
			result = WIN;
		else if (winner == WHITE_WIN && playerColor == PieceColor::White)
			result = WIN;
		else if (winner == DRAW)
			result = DRAW;
		else
			result = LOOSE;

		resultgpu += result;
    }

    printf("Result cpu: %d\n", resultcpu);
    printf("Result gpu: %d\n", resultgpu);
}

int main()
{
    UINT whitePieces = (1ULL << 24) | (1ULL << 26) | (1ULL << 27) | (1ULL << 20) | (1ULL << 16);
    UINT blackPieces = (1ULL << 17) | (1ULL << 18) | (1ULL << 12) | (1ULL << 9) | (1ULL << 7) | (1ULL << 1);
    UINT kings = 0;


    UINT whitePiecesAfter = (1ULL << 24) | (1ULL << 26) | (1ULL << 27) | (1ULL << 20) | (1ULL << 16);
    UINT blackPiecesAfter = (1ULL << 17) | (1ULL << 18) | (1ULL << 12) | (1ULL << 9) | (1ULL << 7) | (1ULL << 4);
	Board board(whitePiecesAfter, blackPiecesAfter, 0);

	// CompareSimulations(board, 1000, PieceColor::Black, PieceColor::Black);

    /*simulateGameGpu(whitePiecesAfter, blackPiecesAfter, 0, PieceColor::Black, 0, 0, 0, 0);
	int result = board.simulateGame(PieceColor::Black);
	printf("Result: %d\n", result);*/

	// CheckersTestSuite::runAll();

    return 0;
}